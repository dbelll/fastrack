#include "hip/hip_runtime.h"
//
//  fastrack.cu
//  fastrack
//
//  Created by Dwight Bell on 12/13/10.
//  Copyright dbelll 2010. All rights reserved.
//

#include <hip/hip_runtime.h>
#include "cutil.h"

#include "fastrack.h"
#include "cuda_utils.h"
#include "rand_utils.h"

static unsigned g_seeds[4] = {0, 0, 0, 0};
static PARAMS g_p;

#pragma mark -
#pragma mark misc.

/*
	Initialize the global seeds using specified seed value.
*/
void set_global_seeds(unsigned seed)
{
	srand(seed);
	for (int i = 0; i < 4; i++) {
		g_seeds[i] = rand();
	}
}

float sigmoid(float x)
{
	return 1.0f/(1.0f + expf(-x));
}

#pragma mark -
#pragma mark allocating and freeing

void freeAgentGPU(AGENT *ag)
{
	if (ag) {
		if (ag->seeds) CUDA_SAFE_CALL(hipFree(ag->seeds));
		if (ag->wgts) CUDA_SAFE_CALL(hipFree(ag->wgts));
		if (ag->W) CUDA_SAFE_CALL(hipFree(ag->W));
		if (ag->alpha) CUDA_SAFE_CALL(hipFree(ag->alpha));
		if (ag->epsilon) CUDA_SAFE_CALL(hipFree(ag->epsilon));
		if (ag->lambda) CUDA_SAFE_CALL(hipFree(ag->lambda));
		CUDA_SAFE_CALL(hipFree(ag));
	}
}

void freeAgentCPU(AGENT *ag)
{
	if (ag) {
		if (ag->seeds) free(ag->seeds);
		if (ag->wgts) free(ag->wgts);
		if (ag->W) free(ag->W);
		if (ag->alpha) free(ag->alpha);
		if (ag->epsilon) free(ag->epsilon);
		if (ag->lambda) free(ag->lambda);
		free(ag);
	}
}

void freeCompactAgent(COMPACT_AGENT *ag)
{
	if (ag) {
		if (ag->seeds) free(ag->seeds);
		if (ag->fdata) free(ag->fdata);
		free(ag);
	}
}


#pragma mark -
#pragma mark game functions

// Calcualte the value for a state s using the specified weights,
// storing hidden activation in the specified location and returning the output value
float val_for_state(float *wgts, int *state, float *hidden)
{
	float out = 0.0f;
	
	for (unsigned iHidden = 0; iHidden < g_p.num_hidden; iHidden++) {
		// first add in the bias
		hidden[iHidden] = -1.0f * wgts[iHidden];
		
		// next, loop over all bits in the state, and add in their contribution
		unsigned iWgt = g_p.num_hidden;	// index into wgts for idx = 0
		for (int i = 0; i < g_p.state_ints; i++) {
			unsigned s = state[i];
			while (s) {
				if (s & 1) hidden[iHidden] += wgts[iHidden + iWgt];
				s >>= 1;
				iWgt += g_p.num_hidden;
			}
		}
		
		// next, apply the activation function
		hidden[iHidden] = sigmoid(hidden[iHidden]);
		
		// now add this hidden node's contribution to the output
		out += hidden[iHidden] * wgts[iHidden + iWgt];
	}
	
	// finally, add the bias to the output value and apply the sigmoid function
	out += -1.0f * wgts[g_p.alloc_wgts - g_p.num_hidden];
	return sigmoid(out);
}

// return a pointer to the starting state for the game
unsigned *start_state()
{
	static unsigned *ss = NULL;
	if (ss == NULL) {
		ss = (unsigned *)calloc(g_p.state_ints, sizeof(unsigned));
		unsigned first_row = (1 << g_p.board_width)-1;
		unsigned last_row = first_row << (32 - g_p.board_width);
		ss[0] = first_row;
		ss[g_p.state_ints-1] |= last_row;
	}
	return ss;
}

#pragma mark -
#pragma mark dump stuff

// Calculate the row value, from the given state for the specified player
unsigned rowbits(unsigned *state, unsigned player, unsigned row)
{
//	printf("rowbits for player %u, row %u\n", player, row);
	unsigned row_bit = player * g_p.board_size + row * g_p.board_width;
//	printf("row_bit is %u\n", row_bit);
	unsigned iState = row_bit / 32;
	unsigned iRowStart = row_bit % 32;
//	printf("iState is %u and iRowStart is %u\n", iState, iRowStart);
	unsigned rb = state[iState] & (((1 << g_p.board_width)-1) << iRowStart);
//	printf("bits before shifting: %u", rb);
	rb >>= row_bit;
//	printf("... bits after shifting: %u\n", rb);
	return rb;
}

void dump_col_header(unsigned leftMargin, unsigned nCols)
{
	while (leftMargin-- > 0) {
		printf(" ");
	}
	for (int i = 0; i < nCols; i++) {
		printf(" %c", 'a' + i);
	}
	printf("\n");
}

void dump_state(unsigned *state)
{
//	printf("dump_state for %u %u %u %u\n", state[3], state[2], state[1], state[0]);
	dump_col_header(3, g_p.board_width);
	for (int r = g_p.board_height - 1; r >= 0; r--) {
		printf("%2u ", r+1);
//		printf("row %d\n", r);
		unsigned s0 = rowbits(state, 0, r);
		unsigned s1 = rowbits(state, 1, r);
//		printf("player 0 row bits is %d\n", s0);
//		printf("player 1 row bits is %d\n", s1);
		unsigned count = 0;
		while (s0 || s1) {
//			printf("s0 = %u  s1 = %u\n", s0, s1);
			if ((s0 & 1) && (s1 & 1)) {
				printf("***ERROR*** inconsistent board\n");
				exit(-1);
			}
			if (s0 & 1) {
				printf(" X");
			}else if (s1 & 1) {
				printf(" O");
			}else {
				printf(" .");
			}
			s0 >>= 1;
			s1 >>= 1;
			count += 1;
		}
		while (count++ < g_p.board_width) {
			printf(" .");
		}
		printf("%3u", r+1);
		printf("\n");
	}
	dump_col_header(3, g_p.board_width);
}


void dump_wgts_header(const char *str)
{
	printf("%s", str);
	for (int i = 0; i < g_p.num_hidden; i++) {
		printf(",  %6d  ", i);
	}
	printf("\n");
}

void dump_wgts(float *wgts)
{
	for (int i = 0; i < g_p.num_hidden; i++) {
		printf(", %9.4f", wgts[i]);
	}
	printf("\n");
}

void dump_agent(AGENT *agCPU, unsigned iag, unsigned dumpW)
{
	printf("[SEEDS], %10d, %10d %10d %10d\n", agCPU->seeds[iag], agCPU->seeds[iag + g_p.num_agents], agCPU->seeds[iag + 2 * g_p.num_agents], agCPU->seeds[iag + 3 * g_p.num_agents]);

	dump_wgts_header("[ WEIGHTS]");
	// get the weight pointer for this agent
	float *pWgts = agCPU->wgts + iag * g_p.alloc_wgts;
	printf("[    B->H]"); dump_wgts(pWgts);
	for (int i = 0; i < g_p.num_inputs; i++){
		printf("[IN%03d->H]", i); dump_wgts(pWgts + (1+i) * g_p.num_hidden);
	}
	printf("[    H->O]"); dump_wgts(pWgts + (1+g_p.num_inputs) * g_p.num_hidden);
	printf("[    B->O], %9.4f\n\n", pWgts[(2+g_p.num_inputs) * g_p.num_hidden]);

	if (dumpW) {
		dump_wgts_header("[    W    ]");
		// get the W pointer for this agent
		float *pW = agCPU->W + iag * g_p.alloc_wgts;
		printf("[    B->H]"); dump_wgts(pW);
		for (int i = 0; i < g_p.num_inputs; i++){
			printf("[IN%03d->H]", i); dump_wgts(pW + (1+i) * g_p.num_hidden);
		}
		printf("[    H->O]"); dump_wgts(pW + (1+g_p.num_inputs) * g_p.num_hidden);
		printf("[    B->O], %9.4f\n\n", pW[(2+g_p.num_inputs) * g_p.num_hidden]);
	}

	printf("[   alpha], %9.4f\n", agCPU->alpha[iag]);
	printf("[ epsilon], %9.4f\n", agCPU->epsilon[iag]);
	printf("[  lambda], %9.4f\n\n", agCPU->lambda[iag]);
}

void dump_agentsCPU(const char *str, AGENT *agCPU, unsigned dumpW)
{
	printf("======================================================================\n");
	printf("%s\n", str);
	printf("----------------------------------------------------------------------\n");
	for (int i = 0; i < g_p.num_agents; i++) {
		printf("[AGENT%5d]\n", i);
		dump_agent(agCPU, i, dumpW);
	}
	printf("======================================================================\n");
	
}

void dump_compact_agent(COMPACT_AGENT *ag)
{
	printf("[SEEDS], %10d, %10d %10d %10d\n", ag->seeds[0], ag->seeds[1], ag->seeds[2], ag->seeds[3]);
	printf("[    B->H]"); dump_wgts(ag->fdata + ag->iWgts);
	for (int i = 0; i < g_p.num_inputs; i++){
		printf("[IN%03d->H]", i); dump_wgts(ag->fdata +ag->iWgts + (1+i) * g_p.num_hidden);
	}
	printf("[    H->O]"); dump_wgts(ag->fdata + ag->iWgts + (1+g_p.num_inputs) * g_p.num_hidden);
	printf("[    B->O], %9.4f\n", ag->fdata[ag->iWgts + (2+g_p.num_inputs) * g_p.num_hidden]);
	printf("[   alpha], %9.4f\n", ag->fdata[ag->iAlpha]);
	printf("[ epsilon], %9.4f\n", ag->fdata[ag->iEpsilon]);
	printf("[  lambda], %9.4f\n", ag->fdata[ag->iLambda]);
}


void dumpResults(RESULTS *r)
{
	printf("Best agents each round...\n");
	for (int i = 0; i < r->allocated; i++) {
		if (r->best+i){
			printf("[ROUND%3d]\n", i);
			dump_compact_agent(r->best+i);
		}
	}
}




#pragma mark -
#pragma mark CPU - Only
RESULTS *newResults()
{
	RESULTS *r = (RESULTS *)malloc(sizeof(RESULTS));
	r->allocated = g_p.num_episodes;
	r->best = (COMPACT_AGENT *)malloc(r->allocated * sizeof(COMPACT_AGENT));
	return r;
}

void freeResults(RESULTS *r)
{
	if (r) {
		for (int i = 0; i < r->allocated; i++)
			if (r->best + i) freeCompactAgent(r->best+i);
	
		free(r);
	}
}

// calculates agent pointers based on offset from ag->wgts
void set_agent_float_pointers(AGENT *ag)
{
	ag->W = ag->wgts + g_p.alloc_wgts * g_p.num_agents;
	ag->alpha = ag->W + g_p.alloc_wgts * g_p.num_agents;
	ag->epsilon = ag->alpha + g_p.num_agents;
	ag->lambda = ag->epsilon + g_p.num_agents;
}

AGENT *init_agentsCPU(PARAMS p)
{
	printf("init_agentsCPU...\n");
	// save the parameters
	g_p = p;

	// allocate and initialize the agent data on CPU
	AGENT *ag = (AGENT *)malloc(sizeof(AGENT));
	unsigned count = 4*p.num_agents;
	ag->seeds = (unsigned *)malloc(count * sizeof(unsigned));
	for (int i = 0; i < 4*p.num_agents; i++) ag->seeds[i] = rand();
	
//	printf("seeds allocated at %p\n", ag->seeds);
	
	// allocate one chunk of float data and set up pointers to appropriate parts of that chunk
	count = (2*p.alloc_wgts + 3) * p.num_agents;
	ag->wgts = (float *)malloc(count * sizeof(float));
	set_agent_float_pointers(ag);
	
//	printf("total of %d float values allocated\n", count);
//	printf("float values allocated at %p, first value is %f\n", ag->wgts, ag->wgts[0]);
//	printf("other pointers ag->W is %p\n", ag->W);
//	printf("           ag->alpha is %p\n", ag->alpha);
//	printf("         ag->epsilon is %p\n", ag->epsilon);
//	printf("          ag->lambda is %p\n", ag->lambda);
	
	// initialize values
	printf("initializing weights for %d values with min of %f and max of %f\n", p.alloc_wgts * p.num_agents, p.init_wgt_min, p.init_wgt_max);
	for (int i=0; i < p.alloc_wgts * p.num_agents; i++){
//		printf("%d ", i);
		ag->wgts[i] = rand_wgt2(p.init_wgt_min, p.init_wgt_max);
//		printf("- ");
		ag->W[i] = 0.0f;
	}
	
	printf("weights and W have been initialized\n");
	
	for (int i = 0; i < p.num_agents; i++) {
		ag->alpha[i] = p.alpha;
		ag->epsilon[i] = p.epsilon;
		ag->lambda[i] = p.lambda;
	}
	
	printf("alpha, epsilon, and lambda have been initialized\n");
	
	return ag;
}

RESULTS *runCPU(AGENT *agCPU)
{
	return NULL;
}



#pragma mark -
#pragma mark GPU - Only

AGENT *init_agentsGPU(AGENT *agCPU)
{
	AGENT *agGPU = (AGENT *)malloc(sizeof(AGENT));
	agGPU->seeds = device_copyui(agCPU->seeds, 4 * g_p.num_agents);
	agGPU->wgts = device_copyf(agCPU->wgts, g_p.agent_float_count * g_p.num_agents);
	set_agent_float_pointers(agGPU);
	
	
	return agGPU;
}


RESULTS *runGPU(AGENT *agGPU)
{
	return NULL;
}




