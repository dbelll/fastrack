#include "hip/hip_runtime.h"
//
//  fastrack.cu
//  fastrack
//
//  Created by Dwight Bell on 12/13/10.
//  Copyright dbelll 2010. All rights reserved.
//

#include <hip/hip_runtime.h>
#include "cutil.h"

#include "fastrack.h"
#include "cuda_utils.h"
#include "rand_utils.h"

//static unsigned g_seeds[4] = {0, 0, 0, 0};
static PARAMS g_p;

// allowable moves expressed as delta from initial position
// maximum distance left/right is board_width/2 - 1
// maximum distance forward/backward is board_width/2 - 1
// {0, 0} is an allowable move, meaning no piece is moved (ie 'pass')
//static int g_allowable_moves[8][2] = {{-1, 2}, {1, 2}, {2, 1}, {2, -1}, {1, -2}, {-1, -2}, {-2, -1}, {-2, 1}};
static int g_allowable_moves[8][2] = MOVES_KNIGHT;

// g_moves will be the move array of size board_size * 8 (for up to 8 possible moves from each cell)
// The values stored is the cell number where the piece would be after the move.  The board_size rows
// in the table are for all the possible starting cell numbers.  A move of -1 is not valid
static int *g_moves = NULL;

static unsigned *g_start_state = NULL;

//static unsigned *g_moves_op = NULL; // used if there is directional difference to possible moves

//static unsigned g_bits_for_board_ints[2];	// stores the number of valid bits in each int that makes up a board

#pragma mark -
#pragma mark misc.

/*
	Initialize the global seeds using specified seed value.
*/
//void set_global_seeds(unsigned seed)
//{
//	srand(seed);
//	for (int i = 0; i < 4; i++) {
//		g_seeds[i] = rand();
//	}
//}

float sigmoid(float x)
{
	return 1.0f/(1.0f + expf(-x));
}

unsigned index4rc(unsigned row, unsigned col)
{
	return row * g_p.board_width + col;
}


#pragma mark -
#pragma mark allocating and freeing

void freeAgentGPU(AGENT *ag)
{
	if (ag) {
		if (ag->seeds) CUDA_SAFE_CALL(hipFree(ag->seeds));
		if (ag->wgts) CUDA_SAFE_CALL(hipFree(ag->wgts));
		if (ag->e) CUDA_SAFE_CALL(hipFree(ag->e));
		if (ag->alpha) CUDA_SAFE_CALL(hipFree(ag->alpha));
		if (ag->epsilon) CUDA_SAFE_CALL(hipFree(ag->epsilon));
		if (ag->lambda) CUDA_SAFE_CALL(hipFree(ag->lambda));
		CUDA_SAFE_CALL(hipFree(ag));
	}
}

void freeAgentCPU(AGENT *ag)
{
	if (ag) {
		if (ag->seeds) free(ag->seeds);
		if (ag->wgts) free(ag->wgts);
		if (ag->e) free(ag->e);
		if (ag->alpha) free(ag->alpha);
		if (ag->epsilon) free(ag->epsilon);
		if (ag->lambda) free(ag->lambda);
		free(ag);
	}
}

void freeCompactAgent(COMPACT_AGENT *ag)
{
	if (ag) {
		if (ag->seeds) free(ag->seeds);
		if (ag->fdata) free(ag->fdata);
		free(ag);
	}
}


#pragma mark -
#pragma mark game functions

unsigned is_empty(unsigned *board)
{
	for (int i = 0; i < g_p.board_size; i++) {
		if (board[i]) return 0;
	}
	return 1;
}

unsigned not_empty(unsigned *board){ return !is_empty(board); }

// Calculate the reward for the given state (from X's perspective)
// non-zero reward ==> terminal state
float reward(unsigned *state, unsigned *terminal)
{
	float reward = 0.0f;
	*terminal = 0;
	if (is_empty(O_BOARD(state))){ reward = 1.0f; *terminal = 1; }
	if (is_empty(X_BOARD(state))){ reward = 0.0f; *terminal = 1; }
	return reward;
}

// Calcualte the value for a state s using the specified weights,
// storing hidden activation in the specified location and returning the output value
float val_for_state(float *wgts, unsigned *state, float *hidden, float *out)
{
//	printf("calculating value for state...\n");
	unsigned terminal;
	float r = reward(state, &terminal);
	if (terminal) return r;

	out[0] = 0.0f;
	
	for (unsigned iHidden = 0; iHidden < g_p.num_hidden; iHidden++) {
		// first add in the bias
		hidden[iHidden] = -1.0f * wgts[iHidden];

		// next loop update for all the input nodes
		for (int i = 0; i < g_p.board_size * 2; i++) {
			if (state[i]) {
				hidden[iHidden] += wgts[iHidden + g_p.num_hidden * (1 + i)];
			}
		}
		
		// apply the sigmoid function
		hidden[iHidden] = sigmoid(hidden[iHidden]);

		// accumulate into the output
		out[0] += hidden[iHidden] * wgts[iHidden + g_p.num_hidden * (1 + g_p.board_size * 2)];
	}
	
	// finally, add the bias to the output value and apply the sigmoid function
	out[0] += -1.0f * wgts[g_p.num_wgts - g_p.num_hidden];
	out[0] = sigmoid(out[0]);
	return out[0];
}

//unsigned int_for_cell(unsigned row, unsigned col)
//{
//	return (col + row * g_p.board_width) / 32;
//}
//
//unsigned bit_for_cell(unsigned row, unsigned col)
//{
//	return (col + row * g_p.board_width) % 32;
//}

unsigned val_for_cell(unsigned row, unsigned col, unsigned *board)
{
	return board[index4rc(row, col)];
}

void set_val_for_cell(unsigned row, unsigned col, unsigned *board, unsigned val)
{
	board[index4rc(row, col)] = val;
}

char char_for_index(unsigned i, unsigned *state)
{
	unsigned s0 = X_BOARD(state)[i];
	unsigned s1 = O_BOARD(state)[i];
	if (s0 && s1) return '?';
	else if (s0) return 'X';
	else if (s1) return 'O';
	return '.';
}

char char_for_cell(unsigned row, unsigned col, unsigned *state)
{
	return char_for_index(index4rc(row, col), state);
}

// add n pieces to un-occupied cells of a board
void random_add(unsigned *board, unsigned n)
{
	while (n > 0) {
		unsigned i = rand() % g_p.board_size;
		if (!board[i]) {
			board[i] = 1;
			--n;
		}
	}
}

// generate a random board
void random_board(unsigned *board, unsigned n)
{
	// first, empty the board
	for (int i = 0; i < g_p.board_size; i++) board[i] = 0;
	
	// now add a random, non-occupied cell
	random_add(board, n);
}


// generate a random board, avoiding any occupied cells in the mask
void random_board_masked(unsigned *board, unsigned *mask, unsigned n)
{
	// first, copy the mask to the board
	for (int i = 0; i < g_p.board_size; i++) board[i] = mask[i];

	// now add a random, non-occupied cell
	random_add(board, n);
	
	// XOR away the mask
	for (int i = 0; i < g_p.board_size; i++) board[i] ^= mask[i];
}

// generate a random state with n pieces for ech player
void random_state(unsigned *state, unsigned n)
{
	random_board(X_BOARD(state), n);
	random_board_masked(O_BOARD(state), X_BOARD(state), n);
}

// return a read-only mask for the specified number of cols on the right
//unsigned *mask_cols_right(unsigned cols)
//{
//	static unsigned need_init = 1;
//	static unsigned *mask;
//	if (need_init) {
//		mask = (unsigned *)calloc(g_p.board_ints * (g_p.board_width-1), sizeof(unsigned));
//		for (int delta = 1; delta < g_p.board_width; delta++) {
//			for (int col = g_p.board_width - delta; col < g_p.board_width; col++) {
//				for (int row = 0; row < g_p.board_height; row++) {
//					set_val_for_cell(row, col, mask + (delta-1)*g_p.board_ints, 1);
//				}
//			}
//		}
//	}
//	return mask + (cols-1)*g_p.board_ints;
//}
//
// return a read-only mask for the specified number of cols on the left
//unsigned *mask_cols_left(unsigned cols)
//{
//	static unsigned need_init = 1;
//	static unsigned *mask;
//	if (need_init) {
//		mask = (unsigned *)calloc(g_p.board_ints * (g_p.board_width-1), sizeof(unsigned));
//		for (int delta = 1; delta < g_p.board_width; delta++) {
//			for (int col = 0; col < delta; col++) {
//				for (int row = 0; row < g_p.board_height; row++) {
//					set_val_for_cell(row, col, mask + (delta-1)*g_p.board_ints, 1);
//				}
//			}
//		}
//	}
//	return mask + (cols-1)*g_p.board_ints;
//}
//
// a mask for the unused bits above the board
//unsigned *mask_rows_top()
//{
//	static unsigned *mask;
//	if (!mask) {
//		mask = (unsigned *)calloc(g_p.board_ints, sizeof(unsigned));
//		if (g_p.board_unused > 0) {
//			mask[g_p.board_ints-1] = - (1 << (32 - g_p.board_unused));
//		}
//	}
//	return mask;
//}
//
// shift the board by a number of columns
//void colshift(unsigned *board, int delta)
//{
//	printf("colshift -- initial board:\n");
//	dump_board(board);
//	
//	if (delta < 0) {
//		delta = -delta;
//
//		printf("shifting the board %d columns to the left\n", delta);
//
//		// shift the low int to the left (lo-bits) first
//		board[0] >>= delta;
//		
//		printf("board after shifting the low int to the left\n");
//		dump_board(board);
//		
//		// get a mask to make the new columns blank
//		unsigned *mask = mask_cols_right(delta);
//
//		if (g_p.board_ints > 1) {
//			// get the bits that will shift out of the hi int
//			unsigned bits = ((1 << delta) - 1) & board[1];
//			// align them to the highest bits
//			bits <<= (32 - delta);
//			// add them to the lo int
//			board[0] |= bits;
//			// now shift the hi int by delta
//			board[1] >>= delta;
//			// mask off the bits in the new cols
//			board[1] &= ~mask[1];
//		}
//		board[0] &= ~mask[0];
//	}else if (delta > 0) {
//	
//		printf("shifting the board %d columns to the right\n", delta);
//
//		// shift the high int to the right (hi-bits) first
//		unsigned *col_mask = mask_cols_left(delta);
//
//		printf("col_mask:\n");
//		dump_board(col_mask);
//
//		unsigned *row_mask = mask_rows_top();
//
//		if (g_p.board_ints > 1) {
//			board[1] <<= delta;
//		
//			printf("board after shifting the hi int to the right\n");
//			dump_board(board);
//		
//			// get the bits that will shift out of the lo int
//			unsigned bits = -(1 << (32-delta)) & board[0];
//			// align them to the lo bits
//			bits >>= 32-delta;
//			// add them to the hi int
//			board[1] |= bits;
//			
//			printf("board after adding the bits that shifted from lo int to hi int\n");
//			dump_board(board);
//			
//			// mask off the rows above the board and cols on the left
//			board[1] &= ((~row_mask[1]) & (~col_mask[1]));
//			
//			printf("board after masking off the hi int\n");
//			dump_board(board);
//		}
//		board[0] <<= delta;
//
//		printf("board after shifting the lo int to the right\n");
//		dump_board(board);
//
//		board[0] &= ~row_mask[0] & ~col_mask[0];
//
//		printf("board after masking off the lo int\n");
//		dump_board(board);
//	}
//}
//
//void rowshift(unsigned *board, int delta)
//{
//	colshift(board, delta * g_p.board_width);
//}
//
//void shift_board(unsigned *board, int colDelta, int rowDelta)
//{
//	colshift(board, colDelta + g_p.board_width * rowDelta);
//}
//
//void shift_state(unsigned *state, int colDelta, int rowDelta)
//{
//	shift_board(state, colDelta, rowDelta);
//	shift_board(state + g_p.board_ints, colDelta, rowDelta);
//}


// copy the starting state to the provided location
void copy_start_state(unsigned *state)
{
	bcopy(g_start_state, state, g_p.state_size * sizeof(unsigned));
}

void switch_sides(unsigned *state)
{
	for (int i = 0; i < g_p.board_size; i++) {
		unsigned temp = X_BOARD(state)[i];
		X_BOARD(state)[i] = O_BOARD(state)[i];
		O_BOARD(state)[i] = temp;
	}
}
void copy_state(unsigned *to, unsigned *from)
{
	for (int i = 0; i < g_p.state_size; i++) {
		to[i] = from[i];
	}
}

unsigned count_pieces(unsigned *board)
{
	unsigned count = 0;
	for (int i = 0; i < g_p.board_size; i++) {
		if (board[i]) ++count;
	}
	return count;
}

#pragma mark -
#pragma mark dump stuff

void dump_col_header(unsigned leftMargin, unsigned nCols)
{
	while (leftMargin-- > 0) {
		printf(" ");
	}
	for (int i = 0; i < nCols; i++) {
		printf(" %c", 'a' + i);
	}
	printf("\n");
}

//void dump_boards(unsigned *b1, unsigned *b2)
//{
//	unsigned *state = (unsigned *)malloc(2 * g_p.state_size * sizeof(unsigned));
//	bcopy(b1, state, g_p.state_size * sizeof(unsigned));
//	bcopy(b2, state + g_p.state_size, g_p.state_size * sizeof(unsigned));
//	dump_state(state);
//}

void dump_state_ints(unsigned *state)
{
	printf("[STATE]\n");
	for (int i = 0; i < g_p.state_size; i++) {
		printf("%11u", state[i]);
	}
	printf("]\n");
}

void dump_state(unsigned *state, unsigned turn, unsigned nextToPlay)
{
//	printf("dump_state for %u %u %u %u\n", state[3], state[2], state[1], state[0]);
	printf("turn %3d, %s to play:\n", turn, (nextToPlay ? "O" : "X"));
	dump_col_header(3, g_p.board_width);
	for (int row = g_p.board_height - 1; row >= 0; row--) {
		printf("%2u ", row+1);
		for (int col = 0; col < g_p.board_width; col++) {
			printf(" %c", char_for_cell(row, col, state));
		}
		printf("%3u", row+1);
		printf("\n");
	}
	dump_col_header(3, g_p.board_width);
//	printf("[STATE");
//	for (int i = 0; i < g_p.state_size; i++) {
//		printf("%11u", state[i]);
//	}
//	printf("\n");
//	dump_state_ints(state);
}

void dump_board(unsigned *board)
{
	dump_col_header(3, g_p.board_width);
	for (int row = g_p.board_height - 1; row >= 0; row--) {
		printf("%2u ", row+1);
		for (int col = 0; col < g_p.board_width; col++) {
			printf(" %c", val_for_cell(row, col, board) ? 'X' : '.');
		}
		printf("%3u", row+1);
		printf("\n");
	}
	dump_col_header(3, g_p.board_width);
}


void dump_wgts_header(const char *str)
{
	printf("%s", str);
	for (int i = 0; i < g_p.num_hidden; i++) {
		printf(",  %6d  ", i);
	}
	printf("\n");
}

void dump_wgts(float *wgts)
{
	for (int i = 0; i < g_p.num_hidden; i++) {
		printf(", %9.4f", wgts[i]);
	}
	printf("\n");
}

void dump_agent(AGENT *agCPU, unsigned iag, unsigned dumpW)
{
	printf("[SEEDS], %10d, %10d %10d %10d\n", agCPU->seeds[iag], agCPU->seeds[iag + g_p.num_agents], agCPU->seeds[iag + 2 * g_p.num_agents], agCPU->seeds[iag + 3 * g_p.num_agents]);

	dump_wgts_header("[ WEIGHTS]");
	// get the weight pointer for this agent
	float *pWgts = agCPU->wgts + iag * g_p.num_wgts;
	printf("[    B->H]"); dump_wgts(pWgts);
	for (int i = 0; i < g_p.state_size; i++){
		printf("[IN%03d->H]", i); dump_wgts(pWgts + (1+i) * g_p.num_hidden);
	}
	printf("[    H->O]"); dump_wgts(pWgts + (1+g_p.state_size) * g_p.num_hidden);
	printf("[    B->O], %9.4f\n\n", pWgts[(2+g_p.state_size) * g_p.num_hidden]);

	if (dumpW) {
		dump_wgts_header("[    W    ]");
		// get the W pointer for this agent
		float *pW = agCPU->e + iag * g_p.num_wgts;
		printf("[    B->H]"); dump_wgts(pW);
		for (int i = 0; i < g_p.state_size; i++){
			printf("[IN%03d->H]", i); dump_wgts(pW + (1+i) * g_p.num_hidden);
		}
		printf("[    H->O]"); dump_wgts(pW + (1+g_p.state_size) * g_p.num_hidden);
		printf("[    B->O], %9.4f\n\n", pW[(2+g_p.state_size) * g_p.num_hidden]);
	}

	printf("[   alpha], %9.4f\n", agCPU->alpha[iag]);
	printf("[ epsilon], %9.4f\n", agCPU->epsilon[iag]);
	printf("[  lambda], %9.4f\n\n", agCPU->lambda[iag]);
}

void dump_agentsCPU(const char *str, AGENT *agCPU, unsigned dumpW)
{
	printf("======================================================================\n");
	printf("%s\n", str);
	printf("----------------------------------------------------------------------\n");
	for (int i = 0; i < g_p.num_agents; i++) {
		printf("[AGENT%5d]\n", i);
		dump_agent(agCPU, i, dumpW);
	}
	printf("======================================================================\n");
	
}

void dump_compact_agent(COMPACT_AGENT *ag)
{
	printf("[SEEDS], %10d, %10d %10d %10d\n", ag->seeds[0], ag->seeds[1], ag->seeds[2], ag->seeds[3]);
	printf("[    B->H]"); dump_wgts(ag->fdata + ag->iWgts);
	for (int i = 0; i < g_p.state_size; i++){
		printf("[IN%03d->H]", i); dump_wgts(ag->fdata +ag->iWgts + (1+i) * g_p.num_hidden);
	}
	printf("[    H->O]"); dump_wgts(ag->fdata + ag->iWgts + (1+g_p.state_size) * g_p.num_hidden);
	printf("[    B->O], %9.4f\n", ag->fdata[ag->iWgts + (2+g_p.state_size) * g_p.num_hidden]);
	printf("[   alpha], %9.4f\n", ag->fdata[ag->iAlpha]);
	printf("[ epsilon], %9.4f\n", ag->fdata[ag->iEpsilon]);
	printf("[  lambda], %9.4f\n", ag->fdata[ag->iLambda]);
}


void dumpResults(RESULTS *row)
{
	printf("Best agents each round...\n");
	for (int i = 0; i < row->allocated; i++) {
		if (row->best+i){
			printf("[ROUND%3d]\n", i);
			dump_compact_agent(row->best+i);
		}
	}
}


#pragma mark -
#pragma mark CPU - setup
RESULTS *newResults()
{
	RESULTS *row = (RESULTS *)malloc(sizeof(RESULTS));
	row->allocated = g_p.num_episodes;
	row->best = (COMPACT_AGENT *)malloc(row->allocated * sizeof(COMPACT_AGENT));
	return row;
}

void freeResults(RESULTS *row)
{
	if (row) {
		for (int i = 0; i < row->allocated; i++)
			if (row->best + i) freeCompactAgent(row->best+i);
	
		free(row);
	}
}

// set the global values for number of valid bits in each board int
//void calc_bits_for_board_ints()
//{
//	g_bits_for_board_ints[0] = (g_p.board_size > 32) ? 32 : g_p.board_size;
//	if (g_p.board_ints == 1) {
//		g_bits_for_board_ints[1] = 0;
//	}else {
//		g_bits_for_board_ints[1] = g_p.board_size - 32;
//	}
//}

// calculates agent pointers based on offset from ag->wgts
// agent data is organized as follows
void set_agent_float_pointers(AGENT *ag)
{
	ag->e = ag->wgts + g_p.num_wgts * g_p.num_agents;
	ag->alpha = ag->e + g_p.num_wgts * g_p.num_agents;
	ag->epsilon = ag->alpha + g_p.num_agents;
	ag->lambda = ag->epsilon + g_p.num_agents;
}


void build_start_state()
{
	g_start_state = (unsigned *)calloc(2*g_p.board_size, sizeof(unsigned));
	for (int i = 0; i < g_p.board_width; i++) {
		X_BOARD(g_start_state)[i] = 1;
		O_BOARD(g_start_state)[(g_p.board_size - 1) - i] = 1;
	}
}

/*
	Build the move array, g_moves, using the allowable moves in g_allowable_moves[8][2]
	g_moves will be of size board_size * 8 * board_size
*/
void build_move_array()
{
	g_moves = (int *)malloc(g_p.board_size * MAX_MOVES * sizeof(int));
	for (int row = 0; row < g_p.board_height; row++) {
		for (int col = 0; col < g_p.board_width; col++) {
			for (int m = 0; m < 8; m++) {
				unsigned iMoves = index4rc(row, col) * MAX_MOVES + m;				
				int toCol = col + g_allowable_moves[m][0];
				int toRow = row + g_allowable_moves[m][1];
				
				if (toCol >= 0 && toCol < g_p.board_width && toRow >= 0 && toRow < g_p.board_height) {
					g_moves[iMoves] = index4rc(toRow, toCol);
				}else {
					g_moves[iMoves] = -1;
				}
			}
		}
	}
}

AGENT *init_agentsCPU(PARAMS p)
{
	printf("init_agentsCPU...\n");
	// save the parameters and calculate any global constants based on the parameters
	g_p = p;
	build_move_array();
	build_start_state();

	// allocate and initialize the agent data on CPU
	AGENT *ag = (AGENT *)malloc(sizeof(AGENT));

	ag->seeds = (unsigned *)malloc(4 * p.num_agents * sizeof(unsigned));
	for (int i = 0; i < 4*p.num_agents; i++) ag->seeds[i] = rand();
	
	ag->wgts = (float *)malloc(p.num_wgts * p.num_agents * sizeof(float));
	ag->e = (float *)malloc(p.num_wgts * p.num_agents * sizeof(float));
	ag->alpha = (float *)malloc(p.num_agents * sizeof(float));
	ag->epsilon = (float *)malloc(p.num_agents * sizeof(float));
	ag->lambda = (float *)malloc(p.num_agents * sizeof(float));
		
	// initialize values
//	printf("initializing weights for %d values with min of %f and max of %f\n", p.num_wgts * p.num_agents, p.init_wgt_min, p.init_wgt_max);
	for (int i=0; i < p.num_wgts * p.num_agents; i++){
		ag->wgts[i] = rand_wgt2(p.init_wgt_min, p.init_wgt_max);
		ag->e[i] = 0.0f;
	}
	
//	printf("weights and W have been initialized\n");
	
	for (int i = 0; i < p.num_agents; i++) {
		ag->alpha[i] = p.alpha;
		ag->epsilon[i] = p.epsilon;
		ag->lambda[i] = p.lambda;
	}
	
//	printf("alpha, epsilon, and lambda have been initialized\n");
	
	return ag;
}

char *move_string(char *buff, unsigned col, unsigned row)
{
	buff[0] = 'a' + col;
	buff[1] = '1' + row;
	return buff;
}

char *move_stringi(char *buff, unsigned i)
{
	return move_string(buff, i % g_p.board_width, i / g_p.board_width);
}



#pragma mark CPU - run

float random_move(unsigned *state)
{
//	printf("choose_move...\n");
	static unsigned *possible_moves = NULL;
	static unsigned allocated = 0;
	if (!possible_moves){
		allocated = 100;
		possible_moves = (unsigned *)malloc(allocated * 2 * sizeof(unsigned));
	}
	unsigned move_count = 0;
	
	// loop through all the possible piece positions
	for (int iFrom = 0; iFrom < g_p.board_size; iFrom++) {
		if (X_BOARD(state)[iFrom]) {
			// found a piece that might be able to move, loop over all possible moves
//			printf("found a piece that can move!\n");
			for (int m = 0; m < MAX_MOVES; m++) {
				int iTo = g_moves[iFrom * MAX_MOVES + m];
				if (iTo >= 0 && !X_BOARD(state)[iTo]) {
					// found a possible move, save it
					if (move_count == allocated) {
						// need to grow the possible moves list
						allocated *= 2;
						possible_moves = (unsigned *)realloc(possible_moves, allocated * 2 * sizeof(unsigned));
					}
					possible_moves[move_count * 2] = iFrom;
					possible_moves[move_count * 2 + 1] = iTo;
					++move_count;
				}
			}
		}
	}
	
	unsigned r = move_count * ranf();
	unsigned iRandFrom = possible_moves[r*2];
	unsigned iRandTo = possible_moves[r*2 + 1];
	// do the random move and return the value
	X_BOARD(state)[iRandFrom] = 0;
	X_BOARD(state)[iRandTo] = 1;
	O_BOARD(state)[iRandTo] = 0;
	
//	printf("best move with value %9.4f:\n", bestVal);
//	dump_state(state);
//	printf("\n\n");
	// recalculate to fill in hidden and out for the chosen move
	return 0.0f;
}


/*
	Choose the move for player X from the given state using the nn specified by wgts.
	Return the value of the best state and over-write currState with the best state.
	calculate the values for every possible next state, and put the best one into nextState, returning its value
*/
float choose_move(unsigned *state, float *wgts, float *hidden, float *out)
{
//	printf("choose_move...\n");
	
	unsigned terminal;
	float r = reward(state, &terminal);
	if (terminal) {
		return r;
	}
	unsigned noVal = 1;
	float bestVal = 0.0f;
	unsigned iBestFrom = 0;
	unsigned iBestTo = 0;
	unsigned move_count = 0;
	
	// loop through all the possible piece positions
	for (int iFrom = 0; iFrom < g_p.board_size; iFrom++) {
		if (X_BOARD(state)[iFrom]) {
			// found a piece that might be able to move, loop over all possible moves
//			printf("found a piece that can move!\n");
			for (int m = 0; m < MAX_MOVES; m++) {
				int iTo = g_moves[iFrom * MAX_MOVES + m];
				if (iTo >= 0 && !X_BOARD(state)[iTo]) {
					// found a possible move, modify the board and calculate the value
					++move_count;
					unsigned oPiece = O_BOARD(state)[iTo];	// remember if there was an O here
					X_BOARD(state)[iFrom] = 0;
					X_BOARD(state)[iTo] = 1;
					O_BOARD(state)[iTo] = 0;
					float val = val_for_state(wgts, state, hidden, out);
//					printf("possible move with value %9.4f:\n", val);
//					dump_state(state);
					if (noVal || val > bestVal) {
//						printf("Best so far !!!\n");
						// record the best move so far
						iBestFrom = iFrom;
						iBestTo = iTo;
						bestVal = val;
						noVal = 0;
					}
					// restore the state
					X_BOARD(state)[iFrom] = 1;
					X_BOARD(state)[iTo] = 0;
					O_BOARD(state)[iTo] = oPiece;
				}
			}
		}
	}
	// do the best move and return the value
	X_BOARD(state)[iBestFrom] = 0;
	X_BOARD(state)[iBestTo] = 1;
	O_BOARD(state)[iBestTo] = 0;
	
//	printf("best move with value %9.4f:\n", bestVal);
//	dump_state(state);
//	printf("\n\n");
	// recalculate to fill in hidden and out for the chosen move
	return val_for_state(wgts, state, hidden, out);
}

// O take an action from the specified state, returning the reward
float take_action(unsigned *state, float *owgts, float *hidden, float *out, unsigned *terminal)
{
	float r = reward(state, terminal);
	if (*terminal) return r;	// given state is terminal, just return the reward
	switch_sides(state);
//	printf("state after switching sides:\n");
//	dump_state(state);
	choose_move(state, owgts, hidden, out);
//	printf("state after opponent move:\n");
//	dump_state(state);
	switch_sides(state);
//	printf("state after switching sides again:\n");
//	dump_state(state);
	return reward(state, terminal);
}

// O takes a random action from the specified state, returning X's reward
float take_random_action(unsigned *state, unsigned *terminal)
{
	float r = reward(state, terminal);
	if (*terminal) return r;	// given state is terminal, just return the reward
	switch_sides(state);
//	printf("state after switching sides:\n");
//	dump_state(state);
	random_move(state);
//	printf("state after opponent move:\n");
//	dump_state(state);
	switch_sides(state);
//	printf("state after switching sides again:\n");
//	dump_state(state);
	return reward(state, terminal);
}

void set_start_state(unsigned *state, unsigned pieces)
{
//	printf("set_start_state...\n");
	(pieces == 0) ? copy_start_state(state) : random_state(state, pieces);
}

void reset_trace(float *e)
{
	for (int i = 0; i < g_p.num_wgts; i++) {
		e[i] = 0.0f;
	}
}

void update_wgts(float alpha, float delta, float *wgts, float *e)
{
	for (int i = 0; i < g_p.num_wgts; i++) {
		wgts[i] += alpha * delta * e[i];
	}
}

// update eligibility traces using the activation values for hidden and output nodes
void update_trace(unsigned *state, float *wgts, float *e, float *hidden, float *out, float lambda)
{
#ifdef DUMP_MOVES
	printf("update_trace\n");
#endif
	// first decay all existing values
	for (int i = 0; i < g_p.num_wgts; i++) {
		e[i] *= g_p.gamma * lambda;
	}
	
	// next update the weights from hidden layer to output node
	// first the bias
	float g_prime_i = out[0] * (1.0f - out[0]);
//	printf("out[0] is %9.4f and g_prime(out) is %9.4f\n", out[0], g_prime_i);
	unsigned iH2O = (2 * g_p.board_size + 1) * g_p.num_hidden;
	e[iH2O + g_p.num_hidden] += -1.0f * g_prime_i;
	
	// next do all the hidden nodes to output node
	for (int j = 0; j < g_p.num_hidden; j++) {
		e[iH2O + j] += hidden[j] * g_prime_i;
//		printf("hidden node %d, activation is %9.4f, increment to e is %9.4f, new e is %9.4f\n", j, hidden[j], g_prime_i*hidden[j], e[iH2O + j]);
	}
	
	// now update the weights to the hidden nodes
	for (int j = 0; j < g_p.num_hidden; j++) {
		float g_prime_j = hidden[j]*(1.0f - hidden[j]) * wgts[iH2O + j] * g_prime_i;
		// first the bias to the hidden node
		e[j] += -1.0f * g_prime_j;
		
		// then all the input -> hidden values
		for (int k = 0; k < g_p.board_size * 2; k++) {
			if (state[k]) e[(k+1)*g_p.num_hidden + j] += g_prime_j;
		}
	}
}

#define SHOW if(show) 

COMPETE_RESULTS compete(float *ag1_wgts, const char *name1, float *ag2_wgts, const char *name2, unsigned start_pieces, unsigned num_games, unsigned turns_per_game, unsigned show)
{
//	printf("\n=================================================================\n");
//	printf("         %s     vs.     %s\n", name1, name2);
	// play a numbr of games, ag1_wgts vs ag2_wgts
	// set up the starting state
	unsigned *state = (unsigned *)malloc(g_p.state_size * sizeof(unsigned));
	float *hidden = (float *)malloc(g_p.num_hidden * sizeof(float));
	float *out = (float *)malloc(g_p.num_hidden * sizeof(float));
	
	COMPETE_RESULTS cr;
	cr.games = num_games;
	cr.wins = 0;
	cr.losses = 0;
	unsigned turn = 0;
	unsigned game = 0;
	unsigned terminal;
	
	SHOW printf(  "-----------------------------------------------------------------\n");
	SHOW printf("game %d:\n", game);

	set_start_state(state, start_pieces);
	if (ranf() < 0.50f) {
		SHOW printf("New game, O plays first\n");
		SHOW dump_state(state, turn, 1);
		(ag2_wgts	? take_action(state, ag2_wgts, hidden, out, &terminal)
					: take_random_action(state, &terminal));
		++turn;
	}else {
		SHOW printf("New game, X plays first\n");
	}
	SHOW dump_state(state, turn, 0);

	
	float V = (ag1_wgts	? choose_move(state, ag1_wgts, hidden, out)
					: random_move(state));
	
	while (game < num_games) {
		SHOW dump_state(state, turn, 1);
		float reward = (ag2_wgts	? take_action(state, ag2_wgts, hidden, out, &terminal)
									: take_random_action(state, &terminal));
		++turn;
		SHOW dump_state(state, turn, 0);
		
		if (terminal){
			if (reward > 0.0f){ ++cr.wins; SHOW printf("*** game won ***\n");}
			if (reward == 0.0f){ ++cr.losses; SHOW printf("*** game lost ***\n");}
		}
		
		if (terminal || (turn == turns_per_game)) {
			SHOW if (!terminal) printf("*** turn limit reached ***\n");
			if (++game < num_games){
				// get ready for next game
				SHOW printf(  "-----------------------------------------------------------------\n");
				SHOW printf("\ngame %d:\n", game);
				turn = 0;
				terminal = 0;
				set_start_state(state, start_pieces);
				if (ranf() < 0.50f) {
					SHOW printf("New game, O plays first\n");
					SHOW dump_state(state, turn, 1);
					(ag2_wgts	? take_action(state, ag2_wgts, hidden, out, &terminal)
								: take_random_action(state, &terminal));
					++turn;
					SHOW dump_state(state, turn, 0);
				}else {
					SHOW printf("New game, X plays first\n");
				}
				SHOW dump_state(state, turn, 0);
			}
		}

		float V_prime = (ag1_wgts	? choose_move(state, ag1_wgts, hidden, out)
								: random_move(state));
		V = V_prime;
	}
	printf("[COMPETE],%5d, %8s,%5d, %8s,%5d", num_games, name1, cr.wins, name2, cr.losses);
//	printf("%7d, %7d, %7d\n", wins, losses, wins-losses);
	free(state);
	free(hidden);
	free(out);
	return cr;
}


// run a learning session using agent ag1 against ag2.  ag2 may be NULL which represents a random player
// Start with a random board with start_pieces per side, or the normal starting board if start_pieces is 0
// Top the learning after num_turns turns (for each player)
void auto_learn(AGENT *ag1, float *ag2_wgts, unsigned start_pieces, unsigned num_games, unsigned turns_per_game)
{
	if (!ag1) {
		printf("***ERROR *** random agent can not learn!!!\n");
		exit(-1);
	}
#ifdef DUMP_ALL_AGENT_UPDATES
	dump_agent(ag1, 0, 1);
#endif
	
#ifdef DUMP_MOVES
	printf("auto_learning for %d games, %d turns per game...\n", num_games, turns_per_game);
#endif	
	// set up the starting state
	unsigned *state = (unsigned *)malloc(g_p.state_size * sizeof(unsigned));
	float *hidden = (float *)malloc(g_p.num_hidden * sizeof(float));
	float *out = (float *)malloc(g_p.num_hidden * sizeof(float));
	
	unsigned wins = 0;
	unsigned losses = 0;
	unsigned turn = 0;
	unsigned terminal = 0;
	unsigned game = 0;
	
	set_start_state(state, start_pieces);
	if (ranf() < 0.50f) {
#ifdef DUMP_MOVES
		printf("New game, O to play first...\n");
		dump_state(state, turn, 1);		
#endif
		float r = (ag2_wgts	? take_action(state, ag2_wgts, hidden, out, &terminal) 
							: take_random_action(state, &terminal));
		++turn;
	}else {
#ifdef DUMP_MOVES
		printf("New game, X to play first...\n");
#endif
	}

#ifdef DUMP_MOVES
	dump_state(state, turn, 0);		
#endif

	// choose the action, storing the next state in agCPU->state and returning the value for the next state
	float V = choose_move(state, ag1->wgts, hidden, out);
	
	update_trace(state, ag1->wgts, ag1->e, hidden, out, ag1->lambda[0]);

#ifdef DUMP_ALL_AGENT_UPDATES
	printf("after updating trace...\n");
	dump_agent(ag1, 0, 1);
#endif
	
	// loop over the number of turns
	while (game < num_games) {

#ifdef DUMP_MOVES
		dump_state(state, turn, 1);
#endif

//		printf("\n\n------- %d turns left -------\n", num_turns+1);
//		printf("after own move, turn %d:\n", turn);
//		dump_state(state);		

#ifdef DUMP_ALL_AGENT_UPDATES
		printf("hidden activation values are:\n");
		for (int i = 0; i < g_p.num_hidden; i++) {
			printf(i == 0 ? "%9.4f" : ", %9.4f", hidden[i]);
		}
		printf("\n");
		printf("output value is %9.4f\n", out[0]);
#endif

		

		float reward = (ag2_wgts	? take_action(state, ag2_wgts, hidden, out, &terminal) 
									: take_random_action(state, &terminal));
		++turn;
#ifdef DUMP_MOVES
		dump_state(state, turn, 0);		
#endif
		
//		printf("after opponent move, turn %d:\n", turn++);
//		dump_state(state);
		
		if (terminal){
			if (reward > 0.0f) ++wins;
			if (reward == 0.0f) ++losses;
#ifdef DUMP_MOVES
			printf("\n\n****** GAME OVER after %d turns with r = %9.4f *******\n", turn, reward);
			printf("record is now %d - %d\n\n\n", wins, losses);
#endif
		}
		
		if (terminal || (turn == turns_per_game)) {
#ifdef DUMP_MOVES
			if (!terminal) printf("****** GAME OVER: reached maximum number of turns per game\n");
#endif
			if (++game < num_games) {
#ifdef DUMP_MOVES
				printf("\n\n--------------- game %d ---------------------\n", game);
#endif
				turn = 0;
				set_start_state(state, start_pieces);
				if (ranf() < 0.50f) {
#ifdef DUMP_MOVES
					printf("New game, O to play first...\n");
					dump_state(state, turn, 1);		
#endif
					(ag2_wgts	? take_action(state, ag2_wgts, hidden, out, &terminal) 
								: take_random_action(state, &terminal));
					++turn;
				}else {
#ifdef DUMP_MOVES
					printf("New game, X to play first...\n");
#endif
				}

#ifdef DUMP_MOVES
				dump_state(state, turn, 0);		
#endif
			}
		}
//		printf("choosing next move...\n");
		float V_prime = choose_move(state, ag1->wgts, hidden, out);
		float delta = reward + (terminal ? 0.0f : (g_p.gamma * V_prime)) - V;
//		printf("updating wgts...\n");
		update_wgts(ag1->alpha[0], delta, ag1->wgts, ag1->e);

#ifdef DUMP_ALL_AGENT_UPDATES
		printf("reward = %9.4f, V_prime = %9.4f, V = %9.4f, delta = %9.4f\n", reward, V_prime, V, delta);
		printf("after updating weights:\n");
		dump_agent(ag1, 0, 1);
#endif

		if (terminal) reset_trace(ag1->e);
//		printf("updating trace...\n");
		update_trace(state, ag1->wgts, ag1->e, hidden, out, ag1->lambda[0]);

#ifdef DUMP_ALL_AGENT_UPDATES
		printf("after updating trace:\n");
		dump_agent(ag1, 0, 1);
#endif
		
		V = V_prime;
	}
//	printf("learning session is over...\n");
//	printf("%7d, %7d, %7d\n", wins, losses, wins-losses);
	free(state);
	free(hidden);
	free(out);
}

void copy_agent(AGENT *agCPU, unsigned iFrom, unsigned iTo)
{
	// copy wgts
//	printf("copying agent weights from %d to %d ... ", iFrom, iTo);
	for (int i = 0; i < g_p.num_wgts; i++) {
		agCPU->wgts[i + iTo * g_p.num_wgts] = agCPU->wgts[i + iFrom * g_p.num_wgts];
	}
//	printf("done\n");
}

void progress_indicator(COMPETE_RESULTS after, COMPETE_RESULTS before)
{
//	printf("\nbefore: W%4d  L%4d  Net%4d", before.wins, before.losses, before.wins - before.losses);
//	printf(" after: W%4d  L%4d  Net%4d", after.wins, after.losses, after.wins - after.losses);
	printf("  ,  W:%+4d L:%+4d NET:%+4d", after.wins - before.wins, after.losses - before.losses, (after.wins - after.losses) - (before.wins - before.losses));
}

#define NAME_BUFF_SIZE 16
const char *aname(unsigned n)
{
	static char buff[NAME_BUFF_SIZE];
	snprintf(buff, NAME_BUFF_SIZE, "FT_%d", n);
	return buff;
}

const char *oname(unsigned n)
{
	static char buff[NAME_BUFF_SIZE];
	snprintf(buff, NAME_BUFF_SIZE, "FT_%d", n);
	return buff;
}

RESULTS *runCPU(AGENT *agCPU)
{	
	// test with agent 0
//	dump_agent(agCPU, 0, 0);

	unsigned pieces = 5;
	unsigned max_turns = 20;
	unsigned games_per_rep = 100;
	unsigned reps = 2000;
	if (reps > g_p.num_agents) reps = g_p.num_agents;

	unsigned test_pieces = pieces;
	unsigned test_games = 10;
	unsigned test_max_turns = 20;
	
	COMPETE_RESULTS *results = (COMPETE_RESULTS *)malloc(reps * sizeof(COMPETE_RESULTS));
	
	printf("NODES, BOARD_WIDTH, BOARD_HEIGHT, PIECES, MAX_TURNS, GAMES, REPS\n");
	printf("%d, %d, %d, %d, %d, %d, %d\n", g_p.num_hidden, g_p.board_width, g_p.board_height, pieces, max_turns, games_per_rep, reps);
//	printf("initial agent, before learning:\n");
//	dump_agent(agCPU, 1, 0);
	

	// get a new set of agents and re-run with lambda = 0.0f
	char *base_name = "20TURNS";
	g_p.alpha = 0.20;
	char *mod_name = "10TURNS";
	AGENT *agCPU2 = init_agentsCPU(g_p);


	COMPETE_RESULTS cr;
	COMPETE_RESULTS totcr;

	// first learn against the random agent...
	auto_learn(agCPU, NULL, pieces, games_per_rep, max_turns);
	
	for (int i = 1; i < reps; i++) {
//		totcr.wins = totcr.losses = 0;
		// save the current state of learning agent after i learning sessions into agent slot i [1..reps-1]
		copy_agent(agCPU, 0, i);
//		printf("agent after %d episodes of learning:\n", i);
//		dump_agent(agCPU, i, 0);

		// do a round of competition, first agent vs. RAND
//		printf("[ROUND%3d], ", i);
//		cr = compete(agCPU->wgts, aname(i), NULL, "RAND", test_pieces, test_games, test_max_turns, 0);
//		// print the progress indicator if agent has competed against this agent before
//		if (i > 1){
//			progress_indicator(cr, results[0]);
//			totcr.wins += cr.wins - results[0].wins;
//			totcr.losses += cr.losses - results[0].losses;
//		}
//		printf("\n");
//		
//		// save the results against this agent
//		results[0] = cr;
//
//		// repeat for all the saved agents, not including the one just saved (at i)
//		for (int j = 1; j < i; j++) {
//			printf("[ROUND%3d], ", i);
//			cr = compete(agCPU->wgts, aname(i), agCPU->wgts + j * g_p.num_wgts, oname(j), test_pieces, test_games, test_max_turns, 0);
////			printf("\n            ");
////			cr = compete(agCPU->wgts + j*g_p.num_wgts, oname(j), agCPU->wgts, aname(i), test_pieces, test_games, test_max_turns, 0);
//			if (j < (i-1)){
//				progress_indicator(cr, results[j]);
//				totcr.wins += cr.wins - results[j].wins;
//				totcr.losses += cr.losses - results[j].losses;
//			}
//			printf("\n");
//			results[j] = cr;
//		}
//		printf("                                              total progress is W:%+4d L:%+4d NET:%+4d\n", totcr.wins, totcr.losses, totcr.wins - totcr.losses);
		// do the next round of learning against the last saved agent (at i)
		auto_learn(agCPU, agCPU->wgts + i * g_p.num_wgts, pieces, games_per_rep, max_turns);
	}
	
	// last round of competition
//	printf("[ROUND%3d], ", reps);
//	cr = compete(agCPU->wgts, aname(reps), NULL, "RAND", test_pieces, test_games, test_max_turns, 0);
//	if (reps > 1) progress_indicator(cr, results[0]);
//	printf("\n");
//	results[0] = cr;
	
//	for (int j = 1; j < reps; j++) {
//		printf("[ROUND%3d], ", reps);
//		cr = compete(agCPU->wgts, aname(reps), agCPU->wgts + j * g_p.num_wgts, oname(j), test_pieces, test_games, test_max_turns, 0);
//		printf("\n            ");
//		cr = compete(agCPU->wgts + j*g_p.num_wgts, oname(j), agCPU->wgts, aname(reps), test_pieces, test_games, test_max_turns, 0);
//		if (j < (reps-1)) progress_indicator(cr, results[j]);
//		printf("\n");
//		results[j] = cr;
//	}



	max_turns = 40;



	// first learn against the random agent...
	auto_learn(agCPU2, NULL, pieces, games_per_rep, max_turns);
	
	for (int i = 1; i < reps; i++) {
//		totcr.wins = totcr.losses = 0;
		// save the current state of learning agent after i learning sessions into agent slot i [1..reps-1]
		copy_agent(agCPU2, 0, i);
//		printf("agent after %d episodes of learning:\n", i);
//		dump_agent(agCPU, i, 0);

//		// do a round of competition, first agent vs. RAND
//		printf("[ROUND%3d], ", i);
//		cr = compete(agCPU2->wgts, aname(i), NULL, "RAND", test_pieces, test_games, test_max_turns, 0);
//		// print the progress indicator if agent has competed against this agent before
//		if (i > 1){
//			progress_indicator(cr, results[0]);
//			totcr.wins += cr.wins - results[0].wins;
//			totcr.losses += cr.losses - results[0].losses;
//		}
//		printf("\n");
//		
//		// save the results against this agent
//		results[0] = cr;
//
//		// repeat for all the saved agents, not including the one just saved (at i)
//		for (int j = 1; j < i; j++) {
//			printf("[ROUND%3d], ", i);
//			cr = compete(agCPU2->wgts, aname(i), agCPU2->wgts + j * g_p.num_wgts, oname(j), test_pieces, test_games, test_max_turns, 0);
//			if (j < (i-1)){
//				progress_indicator(cr, results[j]);
//				totcr.wins += cr.wins - results[j].wins;
//				totcr.losses += cr.losses - results[j].losses;
//			}
//			printf("\n");
//			results[j] = cr;
//		}
//		printf("                                              total progress is W:%+4d L:%+4d NET:%+4d\n", totcr.wins, totcr.losses, totcr.wins - totcr.losses);
//		// do the next round of learning against the last saved agent (at i)
		auto_learn(agCPU2, agCPU2->wgts + i * g_p.num_wgts, pieces, games_per_rep, max_turns);
	}
	
	// last round of competition
//	printf("[ROUND%3d], ", reps);
//	cr = compete(agCPU2->wgts, aname(reps), NULL, "RAND", test_pieces, test_games, test_max_turns, 0);
//	if (reps > 1) progress_indicator(cr, results[0]);
//	printf("\n");
//	results[0] = cr;
//	
//	for (int j = 1; j < reps; j++) {
//		printf("[ROUND%3d], ", reps);
//		cr = compete(agCPU2->wgts, aname(reps), agCPU2->wgts + j * g_p.num_wgts, oname(j), test_pieces, test_games, test_max_turns, 0);
//		if (j < (reps-1)) progress_indicator(cr, results[j]);
//		printf("\n");
//		results[j] = cr;
//	}

	test_games = 1000;
	
	// final competition between the two last agents
	compete(agCPU->wgts, base_name, agCPU2->wgts, mod_name, test_pieces, test_games, test_max_turns, 0);
	printf("\n");
	compete(agCPU2->wgts, mod_name, agCPU->wgts, base_name, test_pieces, test_games, test_max_turns, 0);

	
	// show a few games of against the best saved agent
#ifdef SHOW_SAMPLE_GAMES_AFTER_LEARNING
	compete(agCPU->wgts, base_name, agCPU2->wgts, mod_name, pieces, SHOW_SAMPLE_GAMES_AFTER_LEARNING, max_turns, 1);
#endif
	free(results);
	return NULL;
}


#pragma mark -
#pragma mark GPU - Only

AGENT *init_agentsGPU(AGENT *agCPU)
{
	AGENT *agGPU = (AGENT *)malloc(sizeof(AGENT));
	agGPU->seeds = device_copyui(agCPU->seeds, 4 * g_p.num_agents);
	agGPU->wgts = device_copyf(agCPU->wgts, g_p.num_agent_floats * g_p.num_agents);
	set_agent_float_pointers(agGPU);
	
	return agGPU;
}


RESULTS *runGPU(AGENT *agGPU)
{
	return NULL;
}




